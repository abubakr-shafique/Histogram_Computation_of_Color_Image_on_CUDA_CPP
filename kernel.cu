#include "hip/hip_runtime.h"
//This program is written by Abubakr Shafique (abubakr.shafique@gmail.com) 
#include "hip/hip_runtime.h"
#include ""
#include "CUDA_Histogram.h"

__global__ void Histogram_CUDA(unsigned char* Image, int Channels, int* Histogram_Blue, int* Histogram_Green, int* Histogram_Red);

void Histogram_Calculation_CUDA(unsigned char* Image, int Height, int Width, int Channels, int* Histogram_Blue, int* Histogram_Green, int* Histogram_Red){
	unsigned char* Dev_Image = NULL;
	int* Dev_Histogram_Blue = NULL;
	int* Dev_Histogram_Green = NULL;
	int* Dev_Histogram_Red = NULL;

	//allocate cuda variable memory
	hipMalloc((void**)&Dev_Image, Height * Width * Channels);
	hipMalloc((void**)&Dev_Histogram_Blue, 256 * sizeof(int));
	hipMalloc((void**)&Dev_Histogram_Green, 256 * sizeof(int));
	hipMalloc((void**)&Dev_Histogram_Red, 256 * sizeof(int));

	//copy CPU data to GPU
	hipMemcpy(Dev_Image, Image, Height * Width * Channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_Histogram_Blue, Histogram_Blue, 256 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Dev_Histogram_Green, Histogram_Green, 256 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Dev_Histogram_Red, Histogram_Red, 256 * sizeof(int), hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Histogram_CUDA << <Grid_Image, 1 >> >(Dev_Image, Channels, Dev_Histogram_Blue, Dev_Histogram_Green, Dev_Histogram_Red);

	//copy memory back to CPU from GPU
	hipMemcpy(Histogram_Blue, Dev_Histogram_Blue, 256 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Histogram_Green, Dev_Histogram_Green, 256 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Histogram_Red, Dev_Histogram_Red, 256 * sizeof(int), hipMemcpyDeviceToHost);
	
	//free up the memory of GPU
	hipFree(Dev_Histogram_Blue);
	hipFree(Dev_Histogram_Green);
	hipFree(Dev_Histogram_Red);
	hipFree(Dev_Image);
}

__global__ void Histogram_CUDA(unsigned char* Image, int Channels, int* Histogram_Blue, int* Histogram_Green, int* Histogram_Red){
	int x = blockIdx.x;
	int y = blockIdx.y;

	int Image_Idx = (x + y * gridDim.x) * Channels;

	atomicAdd(&Histogram_Blue[Image[Image_Idx]], 1);
	atomicAdd(&Histogram_Green[Image[Image_Idx + 1]], 1);
	atomicAdd(&Histogram_Red[Image[Image_Idx + 2]], 1);
}
